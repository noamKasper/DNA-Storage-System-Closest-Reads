#include "hip/hip_runtime.h"
#include <iostream>

//const int READ_LENGTH = 115;
const int THREADS_PER_BLOCK = 32;

# if !defined DIVIDE_DATA_BY
# define DIVIDE_DATA_BY 1
# endif

# if !defined UCHAR4_OPTIMIZATION
# define UCHAR4_OPTIMIZATION true
# endif

# if !defined FORCE_UCHAR4_OPTIMIZATION
# define FORCE_UCHAR4_OPTIMIZATION false
# endif

__device__ int editDistance(const char* s, const char* t){
#if (READ_LENGTH <= 115 || !UCHAR4_OPTIMIZATION) && !FORCE_UCHAR4_OPTIMIZATION
    // The last row
    int arr[READ_LENGTH + 1];

    // Initialize arr1 to be the first row of the DP matrix
    for(int j = 0; j <= READ_LENGTH; j++){
        arr[j] = j;
    }

    // Fill the remaining rows
    for(int i = 1; i <= READ_LENGTH; i++) {

        int diag = arr[0];
        arr[0] = i;

        for (int j = 1; j <= READ_LENGTH; j++) {

            int new_val = min(diag + (s[i - 1] != t[j - 1]),
                              min(arr[j] + (s[i - 1] != 'P'), arr[j - 1] + (t[j - 1] != 'P')));
            diag = arr[j];
            arr[j] = new_val;

        }

    }
    return arr[READ_LENGTH];
#else
    uchar4 arr[READ_LENGTH/4 + 1];

    // Initialize arr to be the first row of the DP matrix
    for (int j = 0; j <= READ_LENGTH; j+=4){
        arr[j].x = j;
        arr[j].y = j+1;
        arr[j].z = j+2;
        arr[j].w = j+3;
    }

    // Fill the remaining rows
    for(int i = 1; i <= READ_LENGTH; i++) {

        int diag = arr[0].x;
        arr[0].x = i;

        for (int j = 1; j <= READ_LENGTH; j++) {
            switch (j%4){
                case 0:
                    {
                    // x
                    int new_val = min(diag + (s[i - 1] != t[j - 1]),
                              min(arr[j/4].x + (s[i - 1] != 'P'), arr[j/4 - 1].w + (t[j - 1] != 'P')));
                    diag = arr[j/4].x;
                    arr[j/4].x = new_val;
                    }
                    break;
                case 1:
                    {
                    // y
                    int new_val = min(diag + (s[i - 1] != t[j - 1]),
                              min(arr[j/4].y + (s[i - 1] != 'P'), arr[j/4].x + (t[j - 1] != 'P')));
                    diag = arr[j/4].y;
                    arr[j/4].y = new_val;
                    }
                    break;
                case 2:
                    {
                    // z
                    int new_val = min(diag + (s[i - 1] != t[j - 1]),
                              min(arr[j/4].z + (s[i - 1] != 'P'), arr[j/4].y + (t[j - 1] != 'P')));
                    diag = arr[j/4].z;
                    arr[j/4].z = new_val;
                    }
                    break;
                case 3:
                    {
                    // w
                    int new_val = min(diag + (s[i - 1] != t[j - 1]),
                              min(arr[j/4].w + (s[i - 1] != 'P'), arr[j/4].z + (t[j - 1] != 'P')));
                    diag = arr[j/4].w;
                    arr[j/4].w = new_val;
                    }
                    break;
            }
        }
    }

    int last =  READ_LENGTH/4;
    switch (READ_LENGTH % 4){
        case 0: return arr[last].x;
        case 1: return arr[last].y;
        case 2: return arr[last].z;
        case 3: return arr[last].w;
    }

#endif
}

__global__ void findClosest(const char *reads, int *min_num, int *min_index, int num_reads){

    __shared__ int min_distance;
    __shared__ int minIdx;
    __shared__ char read[READ_LENGTH];

    if (threadIdx.x == 0) {
        min_distance = READ_LENGTH;
        minIdx = -1;
        for (int i = 0; i < READ_LENGTH; i++) read[i] = reads[(READ_LENGTH * blockIdx.x) + i];
    }
    __syncthreads();

    for (int i = 0; i < num_reads; i += THREADS_PER_BLOCK) {
        int index = threadIdx.x + i;
        if (blockIdx.x != index && index < num_reads) {
            int edit_distance = editDistance(read, (reads + READ_LENGTH * index));
            if(edit_distance < min_distance){
                int previous = atomicMin(&min_distance, edit_distance);
                if(edit_distance < previous && edit_distance == min_distance){
                    minIdx = index;
                }
            }
        }
        __syncthreads();
    }

    if(threadIdx.x == 0){
        min_num[blockIdx.x] = min_distance;
        min_index[blockIdx.x] = minIdx;
    }

}

int main(){
//    hipSetDevice(4);

    std::string readsStr;
    std::cin >> readsStr;
    int reads_length = readsStr.length();

    int num_reads = reads_length / READ_LENGTH;
    const char *reads = readsStr.c_str();
    char *d_reads; hipMalloc(&d_reads,reads_length * sizeof(char));
    hipMemcpy(d_reads, reads, reads_length * sizeof(char), hipMemcpyHostToDevice);

    // will be a list of the minimum edit distance for each read
    int *min_num = (int*) malloc(num_reads * sizeof(int));
    int *d_min_num; hipMalloc(&d_min_num, num_reads * sizeof(int));

    // will be a list of the index of the minimum edit distance for each read
    int *min_index = (int*) malloc(num_reads * sizeof(int));
    int *d_min_index; hipMalloc(&d_min_index, num_reads * sizeof(int));
// divide by 4 blocks to split dataset
    findClosest<<<num_reads/DIVIDE_DATA_BY, THREADS_PER_BLOCK>>>(d_reads,d_min_num, d_min_index, num_reads);

    hipMemcpy(min_num, d_min_num, num_reads * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(min_index, d_min_index, num_reads * sizeof(int), hipMemcpyDeviceToHost);

    std::cout<< "settings: " <<-1 << "," <<-1 << "," << -1 << "," << -1 << "," << num_reads << std::endl;
    std::cout << "read index" << ","<< "closest read" << ","<< "edit distance" << std::endl;
    for(int i = 0; i < num_reads; i++){
        std::cout << i << ","<< min_index[i]<< ","<< min_num[i] << std::endl;
    }

    return 0;
}