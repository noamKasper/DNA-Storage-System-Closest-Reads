#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <string.h>

struct Histogram{
    unsigned char numA;
    unsigned char numT;
    unsigned char numC;
    unsigned char numG;
};

const int NUM_READS = 256;
const int N = 256;
const int THREADS_PER_BLOCK = 32;
const int ETH = 200;
const int K = 12;

__device__ int editDistance(const char* s, const char* t){

    // The last row
    int arr[N + 1];

    // Initialize arr to be the first row of the DP matrix
    for(int j = 0; j <= N; j++){
        arr[j] = j;
    }

    // Fill the remaining rows
    for(int i = 1; i <= N; i++){
    
        int diag = arr[0];
        arr[0] = i;

        for(int j = 1; j <= N; j++){

            int new_val = min(diag + (s[i - 1] != t[j - 1]), min(arr[j], arr[j - 1]) + 1);
            diag = arr[j];
            arr[j] = new_val;

        }

    }

    return arr[N];

}

//__device__ const char* decimalToDna(int num) {
//
//    // Stores DNA representation of number.
//    char DnaNum[K];
//    char DNA[5] = "ACGT";
//
//    for (int i = 0 ;i < K;i++ ){
//        DnaNum[i] = DNA[num % 4];
//        num /= 4;
//    }
//    return DnaNum;
//}
__constant__ int RHO[26] = {0, 0, 1, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 0, 0, 0};//possibilities
__device__ int dnaToDecimal(const char* dnaSeq) {
    int decimalNum = 0;
    for (int i = K; i >= 0; i--) {
        int nucleotideValue = RHO[dnaSeq[i] - 'A'];
        decimalNum = decimalNum * 4 + nucleotideValue;
    }
    return decimalNum;
}


__global__ void computeHistogram(const char *reads,Histogram *histograms){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    histograms[index].numA = 0;
    histograms[index].numT = 0;
    histograms[index].numG = 0;
    histograms[index].numC = 0;
    for(int i = 0; i < N; i++){
        histograms[index].numA += (reads[index * N + i] == 'A');
        histograms[index].numT += (reads[index * N + i] == 'T');
        histograms[index].numG += (reads[index * N + i] == 'G');
        histograms[index].numC += (reads[index * N + i] == 'C');
    }
    
}
template <unsigned int n, typename T>
/**
 * A cyclic buffer of size n
 */
struct CyclicBuffer{
    /**
     * The buffer itself
     */
    T buffer[n];
    /**
     * The buffer start location
     */
    unsigned int start;
    /**
     * The buffer length
     */
    unsigned int length;
    /**
     * Resets the buffer
     */
    __device__ void reset(){
        start = 0; length = 0;
    }
    /**
     * Returns the xth element in the buffer
     * @param x
     * @return
     */
    __device__ T get(unsigned int x){
        return buffer[(start + x) % n];
    }
    /**
     * Sets the xth element in the buffer
     * @param x
     * @param data
     */
    __device__ void set(unsigned int x, T data){
        buffer[(start + x) % n] = data;
    }
    /**
     * Enlarges the buffer by numElements. Should only be called by thread 0.
     * @param numElements
     */
    __device__ void push(unsigned int numElements){
        if (n < length + numElements){
            printf("length is bigger then numElements");
//            exit(1);
        }
        if(threadIdx.x == 0) length += numElements;
    }
    /**
     * Shrinks the buffer by numElements. Should only be called by thread 0.
     * @param numElements
     */
    __device__ void pop(unsigned int numElements){
        if(threadIdx.x == 0){
            start = (start + numElements) % n;
            if (length < numElements){
                printf("length is smaller then numElements");
//                exit(1);
            }
            length -= numElements;
        }
    }
};



__global__ void findClosest(const char *reads, int *min_num, int *min_index, Histogram *histograms){

    // initializing variables
    __shared__ CyclicBuffer <2 * THREADS_PER_BLOCK, int> samplesBuffer;
    samplesBuffer.reset();  // resets the buffer
    
    __shared__ int min_distance;
    __shared__ int minIdx;
    __shared__ int count;
    __shared__ char read[N];


    //creates the main read in the memory:
    if (threadIdx.x == 0) {
        min_distance = N;
        minIdx = -1;
        for (int i = 0; i < N; i++) read[i] = reads[(N * blockIdx.x) + i];
    }
    __syncthreads();


    for (int i = 0; i < NUM_READS; i += THREADS_PER_BLOCK){
        if (blockIdx.x != (i + threadIdx.x)) {
            count = 0;
            int length = samplesBuffer.length;  //enables to know the current length
            // Compute histogram diff between our read and read i + threadIdx.x

            int diff =  abs(histograms[blockIdx.x].numA - histograms[i + threadIdx.x].numA) +
                    abs(histograms[blockIdx.x].numT - histograms[i + threadIdx.x].numT) +
                    abs(histograms[blockIdx.x].numG - histograms[i + threadIdx.x].numG) +
                    abs(histograms[blockIdx.x].numC - histograms[i + threadIdx.x].numC);
            // If diff < 2*ETH, add to samples buffer
            if (diff < 2*ETH){
                int readBufferIdx = atomicAdd(&count, 1);  //if I did normal add it would be overwritten, returns count before add
                samplesBuffer.set(readBufferIdx + length, i + threadIdx.x);
            }
        }

        __syncthreads();
        if (threadIdx.x == 0){
            samplesBuffer.push(count);//enlarges the length of the buffer
        }
        __syncthreads();

        if(samplesBuffer.length >= THREADS_PER_BLOCK){
            int edit_distance = editDistance(read, (reads + N * (samplesBuffer.get(threadIdx.x)))); // samplesBuffer[threadIdx.x]
            if(edit_distance < min_distance){
                int previous = atomicMin(&min_distance, edit_distance);
                if(edit_distance < previous && edit_distance == min_distance){
                    minIdx = samplesBuffer.get(threadIdx.x);
                }
            }
            __syncthreads();
            if(threadIdx.x == 0){
                samplesBuffer.pop(THREADS_PER_BLOCK); //removes all of the checked reads
            }
            __syncthreads();
        }


    }
    // empty samplesBuffer one last time
    if(samplesBuffer.length > 0) {
        if (threadIdx.x < samplesBuffer.length){
            int edit_distance = editDistance(read,
                                             (reads + N * (samplesBuffer.get(threadIdx.x)))); // samplesBuffer[threadIdx.x]
            if (edit_distance < min_distance) {
                int previous = atomicMin(&min_distance, edit_distance);
                if (edit_distance < previous && edit_distance == min_distance) {
                    minIdx = samplesBuffer.get(threadIdx.x);
                }
            }
        }
        __syncthreads();

    }
//    printf("The minimum_num of thread %d block %d is: %d\n",threadIdx.x,blockIdx.x,minimum_num);

    if(threadIdx.x == 0){
        min_num[blockIdx.x] = min_distance;
        min_index[blockIdx.x] = minIdx;
    }
    __syncthreads();

}

int main(){

    std::string readsStr;
    std::cin >> readsStr;
    const char *reads = readsStr.c_str();

    char *d_reads; hipMalloc(&d_reads,NUM_READS*sizeof(char)*N);
    hipMemcpy(d_reads, reads, NUM_READS * sizeof(char)*N, hipMemcpyHostToDevice);

//    Histogram *histograms = (Histogram*) malloc(NUM_READS * sizeof(Histogram));
    Histogram *d_histograms; hipMalloc(&d_histograms, NUM_READS * sizeof(Histogram));

    DynamicVector *index_table = (DynamicVector*) malloc(std::pow(4,K) * sizeof(DynamicVector));
    DynamicVector *d_index_table; hipMalloc(&d_index_table, std::pow(4,K) * sizeof(DynamicVector));
    // will be a list of the minimum edit distance for each read
    int *min_num = (int*) malloc(NUM_READS * sizeof(int));
    int *d_min_num; hipMalloc(&d_min_num, NUM_READS * sizeof(int));

    // will be a list of the index of the minimum edit distance for each read
    int *min_index = (int*) malloc(NUM_READS * sizeof(int));
    int *d_min_index; hipMalloc(&d_min_index, NUM_READS * sizeof(int));

//    int *d_read_counts; hipMalloc(&d_read_counts, std::pow(4,K) * sizeof(int));


    computeHistogram<<<NUM_READS/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_reads,d_histograms);


    findClosest<<<NUM_READS,THREADS_PER_BLOCK>>>(d_reads,d_min_num, d_min_index, d_histograms);


    hipMemcpy(min_num, d_min_num, NUM_READS * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(min_index, d_min_index, NUM_READS * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "read index" << ","<< "closest read" << ","<< "edit distance" << std::endl;
    for(int i = 0; i < NUM_READS; i++){
        std::cout << i << ","<< min_index[i]<< ","<< min_num[i] << std::endl;
    }

    return 0;
}