#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <string.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

# if !defined USED_READS_SIZE
# define USED_READS_SIZE 2048
# endif

# if !defined ETH
# define ETH 10
# endif

# if !defined KMER
# define KMER 12
# endif

# if !defined DIVIDE_DATA_BY
# define DIVIDE_DATA_BY 1
# endif

# if !defined UCHAR4_OPTIMIZATION
# define UCHAR4_OPTIMIZATION true
# endif

# if !defined FORCE_UCHAR4_OPTIMIZATION
# define FORCE_UCHAR4_OPTIMIZATION false
# endif

#define NEW_VAL(s_val, t_val) min(diag + (s[i - 1] != t[j - 1]), min(s_val + (s[i - 1] != 'P'),  t_val + (t[j - 1] != 'P')));

const int THREADS_PER_BLOCK = 32;

struct Histogram{
    unsigned char numA;
    unsigned char numT;
    unsigned char numC;
    unsigned char numG;
};

struct IndexTable{
    int index;
    int count;
};

template <unsigned int n, typename T>
/**
 * A cyclic buffer of size n
 */
struct CyclicBuffer{
    /**
     * The buffer itself
     */
    T buffer[n];
    /**
     * The buffer start location
     */
    unsigned int start;
    /**
     * The buffer length
     */
    unsigned int length;
    /**
     * Resets the buffer
     */
    __device__ void reset(){
        start = 0; length = 0;
    }
    /**
     * Returns the xth element in the buffer
     * @param x
     * @return
     */
    __device__ T get(unsigned int x){
        return buffer[(start + x) % n];
    }
    /**
     * Sets the xth element in the buffer
     * @param x
     * @param data
     */
    __device__ void set(unsigned int x, T data){
        buffer[(start + x) % n] = data;
    }
    /**
     * Enlarges the buffer by numElements. Should only be called by thread 0.
     * @param numElements
     */
    __device__ void push(unsigned int numElements){
        if (n <= length + numElements){
            printf("length is bigger then n\n");
            return;
        }
        if(threadIdx.x == 0) length += numElements;
    }
    /**
     * Shrinks the buffer by numElements. Should only be called by thread 0.
     * @param numElements
     */
    __device__ void pop(unsigned int numElements){
        if(threadIdx.x == 0){
            start = (start + numElements) % n;
            if (length < numElements){
                printf("length is smaller then numElements\n");
//                exit(1);
            }
            length -= numElements;
        }
    }
    /**
     * Returns true if buffer contains data.
     * @param data
     */
    __device__ bool contains(T data){
        for(int i = 0; i < length; i++){
            if (get(i) == data){
                return true;
            }
        }
        return false;
    }
};

__device__ int editDistance(const char* s, const char* t){
#if (READ_LENGTH <= 115 || !UCHAR4_OPTIMIZATION) && !FORCE_UCHAR4_OPTIMIZATION
    // The last row
    int arr[READ_LENGTH + 1];

    // Initialize arr1 to be the first row of the DP matrix
    for(int j = 0; j <= READ_LENGTH; j++){
        arr[j] = j;
    }

    // Fill the remaining rows
    for(int i = 1; i <= READ_LENGTH; i++) {

        int diag = arr[0];
        arr[0] = i;

        for (int j = 1; j <= READ_LENGTH; j++) {

            int new_val = min(diag + (s[i - 1] != t[j - 1]),
                              min(arr[j] + (s[i - 1] != 'P'), arr[j - 1] + (t[j - 1] != 'P')));
            diag = arr[j];
            arr[j] = new_val;

        }

    }
    return arr[READ_LENGTH];
#else
    uchar4 arr[READ_LENGTH/4 + 1];

    // Initialize arr to be the first row of the DP matrix
    for (int j = 0; j <= READ_LENGTH; j+=4){
        arr[j/4].x = j;
        arr[j/4].y = j+1;
        arr[j/4].z = j+2;
        arr[j/4].w = j+3;
    }

    // Fill the remaining rows
    for(int i = 1; i <= READ_LENGTH; i++) {

        int diag = arr[0].x;
        arr[0].x = i;

        for (int j = 1; j <= READ_LENGTH; j++) {
            switch (j%4){
                case 0:
                {
                    // x
                    int new_val = NEW_VAL(arr[j/4].x, arr[j/4 - 1].w);
                    diag = arr[j/4].x;
                    arr[j/4].x = new_val;
                }
                    break;
                case 1:
                {
                    // y
                    int new_val = NEW_VAL(arr[j/4].y, arr[j/4].x);
                    diag = arr[j/4].y;
                    arr[j/4].y = new_val;
                }
                    break;
                case 2:
                {
                    // z
                    int new_val = NEW_VAL(arr[j/4].z, arr[j/4].y);
                    diag = arr[j/4].z;
                    arr[j/4].z = new_val;
                }
                    break;
                case 3:
                {
                    // w
                    int new_val = NEW_VAL(arr[j/4].w, arr[j/4].z);
                    diag = arr[j/4].w;
                    arr[j/4].w = new_val;
                }
                    break;
            }
        }
    }

    int last =  READ_LENGTH/4;
    switch (READ_LENGTH % 4){
        case 0: return arr[last].x;
        case 1: return arr[last].y;
        case 2: return arr[last].z;
        case 3: return arr[last].w;
    }

#endif
}


__constant__ int RHO[26] = {0, 0, 1, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 0, 0, 0};//possibilities

/**
 * Calculates the decimal value of a DNA sequence.
 * @param dnaSeq
 */
__device__ int dnaToDecimal(const char* dnaSeq) {
    int decimalNum = 0;
    for (int i = KMER-1; i >= 0; i--) {
        char base = dnaSeq[i];
        if (base < 'A' || base > 'Z'){
            printf("illegal base: %c", base);
            return 0;
        }
        int nucleotideValue = RHO[base - 'A'];
        decimalNum = decimalNum * 4 + nucleotideValue;
    }
    return decimalNum;
}

/**
 * Calculates the base counts for each read.
 * @param reads
 * @param histograms
 * @param num_reads
 */
__global__ void computeHistogram(const char *reads, Histogram *histograms, int num_reads){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_reads) return;
    histograms[index].numA = 0;
    histograms[index].numT = 0;
    histograms[index].numG = 0;
    histograms[index].numC = 0;
    for(int i = 0; i < READ_LENGTH; i++){
        histograms[index].numA += (reads[index * READ_LENGTH + i] == 'A');
        histograms[index].numT += (reads[index * READ_LENGTH + i] == 'T');
        histograms[index].numG += (reads[index * READ_LENGTH + i] == 'G');
        histograms[index].numC += (reads[index * READ_LENGTH + i] == 'C');
    }
}


/**
 * Calculates the number of reads containing each KMER
 * @param reads
 * @param read_counts
 * @param num_reads
 */
__global__ void computeReadCounts(const char *reads, int *read_counts, int num_reads){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_reads) return;
    for (int i = 0; i <= READ_LENGTH - KMER; i++){
        char kmer[KMER];
        for (int j = 0; j < KMER; j++) kmer[j] = reads[(READ_LENGTH * index)+j+i];
        atomicAdd(&read_counts[dnaToDecimal(kmer)], 1);
    }
}


/**
 * initializes an array of IndexTable instances that for each KMER contains the first read in the read chunk and the number of reads containing the KMER
 * @param reads
 * @param read_counts
 * @param tmp_read_counts
 * @param read_chunk
 * @param prefix
 * @param index_table
 * @param num_reads
 */
__global__ void computeIndexTable(const char *reads, int *read_counts, int *tmp_read_counts, int *read_chunk, int *prefix, IndexTable *index_table, int num_reads){
    int read = blockIdx.x * blockDim.x + threadIdx.x;
    if (read >= num_reads) return;
    for (int i = 0; i <= READ_LENGTH - KMER; i++){
        char kmer[KMER];
        for (int j = 0; j < KMER; j++) kmer[j] = reads[(READ_LENGTH * read)+j+i];
        int kmerNum = dnaToDecimal(kmer);
        int value = atomicAdd(&tmp_read_counts[kmerNum], 1);
        int index = prefix[kmerNum] + value;
        read_chunk[index] = read;
        index_table[kmerNum].index = prefix[kmerNum];
        index_table[kmerNum].count = read_counts[kmerNum];
    }
}

__global__ void findClosest(const char *reads, int *min_num, int *min_index, Histogram *histograms, IndexTable *index_table, int *read_chunk, unsigned long long *clock, long *counter_sum){
    unsigned long long start = clock64();
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    long counter = 0;

    // will contain all the potential closest reads
    __shared__ CyclicBuffer <2 * THREADS_PER_BLOCK, int> samplesBuffer;
    samplesBuffer.reset();

    // will contain the first USED_READS_SIZE(number) that were checked with edit distance
    __shared__ CyclicBuffer <USED_READS_SIZE, int> usedReads;
    usedReads.reset();

    __shared__ int min_distance;
    __shared__ int minIdx;
    __shared__ int count;
    __shared__ char read[READ_LENGTH];


    //creates the main read in the memory:
    if (threadIdx.x == 0) {
        min_distance = READ_LENGTH;
        minIdx = -1;
        for (int i = 0; i < READ_LENGTH; i++) read[i] = reads[(READ_LENGTH * blockIdx.x) + i];
    }
    __syncthreads();

    // iterate over each KMER in the block's read
    for (int i = 0; i <= READ_LENGTH - KMER; i++) {
        // copy the KMER to array
        char kmer[KMER];
        for (int j = 0; j < KMER; j++) kmer[j] = read[j + i];

        IndexTable kmer_reads = index_table[dnaToDecimal(kmer)];

        // iterate over each read containing the current KMER
        for (int j = 0; j < kmer_reads.count; j += THREADS_PER_BLOCK) {
            if (j + threadIdx.x < kmer_reads.count) {

                int comp_read = read_chunk[kmer_reads.index + j + threadIdx.x];
                count = 0;
                if (blockIdx.x != comp_read && !usedReads.contains(comp_read)) {
                    int samplesLength = samplesBuffer.length;

                    // calculates base counts diff between block read and another read.
                    int diff = abs(histograms[blockIdx.x].numA - histograms[comp_read].numA) +
                               abs(histograms[blockIdx.x].numT - histograms[comp_read].numT) +
                               abs(histograms[blockIdx.x].numG - histograms[comp_read].numG) +
                               abs(histograms[blockIdx.x].numC - histograms[comp_read].numC);

                    // if diff is smaller the 2*ETH (this is the filter) add it to the buffer
                    if (diff < 2 * ETH) {
                        int readBufferIdx = atomicAdd(&count, 1);
                        samplesBuffer.set(readBufferIdx + samplesLength, comp_read);
                    }
                }
            }
            __syncthreads();

            if (threadIdx.x == 0) {
                samplesBuffer.push(count); //enlarges the length of the buffer
            }
            __syncthreads();

            // if buffer is big enough, check 32 reads from the buffer
            if (samplesBuffer.length >= THREADS_PER_BLOCK) {
                counter++;
                // calculate edit distance
                int edit_distance = editDistance(read, (reads + READ_LENGTH * (samplesBuffer.get(threadIdx.x))));
                if (edit_distance < min_distance) {
                    int previous = atomicMin(&min_distance, edit_distance);
                    // make sure that this thread has the actual smallest edit distance
                    if (edit_distance < previous && edit_distance == min_distance) {
                        minIdx = samplesBuffer.get(threadIdx.x);
                    }
                }

                // removes checked reads and adds them the used reads buffer
                if (threadIdx.x == 0){
                    if (usedReads.length + THREADS_PER_BLOCK < USED_READS_SIZE){
                        usedReads.push(THREADS_PER_BLOCK);
                    }else{
                        usedReads.push(USED_READS_SIZE - usedReads.length -1);
                    }
                }
                int threadLoc = usedReads.length + threadIdx.x;
                __syncthreads();
                if (threadLoc < USED_READS_SIZE){
                    usedReads.set(threadLoc, samplesBuffer.get(threadIdx.x));
                }
                if (threadIdx.x == 0) {
                    samplesBuffer.pop(THREADS_PER_BLOCK);
                }
                __syncthreads();
            }

        }
    }
    // empty samplesBuffer one last time
    if(samplesBuffer.length > 0) {
        if (threadIdx.x < samplesBuffer.length){
            int edit_distance = editDistance(read, (reads + READ_LENGTH * (samplesBuffer.get(threadIdx.x))));
            counter++;

            if (edit_distance < min_distance) {
                int previous = atomicMin(&min_distance, edit_distance);
                if (edit_distance < previous && edit_distance == min_distance) {
                    minIdx = samplesBuffer.get(threadIdx.x);
                }
            }
        }
        __syncthreads();

    }

    if(threadIdx.x == 0){
        min_num[blockIdx.x] = min_distance;
        min_index[blockIdx.x] = minIdx;

    }
    __syncthreads();
    unsigned long long end = clock64();
    atomicAdd(clock, (end - start));
    counter_sum[index] = counter;
}

int main(){

    // receive dataset as input
    std::string readsStr;
    std::cin >> readsStr;
    const char *reads = readsStr.c_str();
    char *d_reads; hipMalloc(&d_reads,reads_length * sizeof(char));
    hipMemcpy(d_reads, reads, reads_length * sizeof(char), hipMemcpyHostToDevice);

    // get file length
    int reads_length = readsStr.length();

    // calculate the number of reads in the dataset
    int num_reads = reads_length / READ_LENGTH;

    // create histogram in the GPU memory
    Histogram *d_histograms; hipMalloc(&d_histograms, num_reads * sizeof(Histogram));

    // initialize clock in CPU and GPU memory
    unsigned long long clock = 0;
    unsigned long long *d_clock; hipMalloc(&d_clock, sizeof(unsigned long long));

    // initialize a counter that counts every edit distance calculated (used for testing purposes only)
    long *counts = (long*) malloc(num_reads * THREADS_PER_BLOCK * sizeof(long));
    long *d_counts; hipMalloc(&d_counts, num_reads * THREADS_PER_BLOCK * sizeof(long));

    // initialize an array of the minimum edit distance for each read
    int *min_num = (int*) malloc(num_reads * sizeof(int));
    int *d_min_num; hipMalloc(&d_min_num, num_reads * sizeof(int));

    // initialize an array of the index of the minimum edit distance for each read
    int *min_index = (int*) malloc(num_reads * sizeof(int));
    int *d_min_index; hipMalloc(&d_min_index, num_reads * sizeof(int));

    // initialize an array of the counts of each KMER
    int *d_read_counts; hipMalloc(&d_read_counts, std::pow(4,KMER) * sizeof(int)); hipMemset(d_read_counts, 0, std::pow(4,KMER) * sizeof(int));

    // initialize an array that sum of all the values before each value in the counts array.
    // will be used to locate each KMER in the chunk array
    int *read_prefix = (int*) malloc(std::pow(4,KMER) * sizeof(int));
    int *d_read_prefix; hipMalloc(&d_read_prefix, std::pow(4,KMER) * sizeof(int));

    // creating a device(GPU) pointer enables to use reduce in the thrust library
    thrust::device_ptr<int> dev_ptr(d_read_counts);

    // calculate the base counts for each read
    computeHistogram<<<num_reads/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_reads,d_histograms, num_reads);

    // calculates the number of reads containing each KMER
    computeReadCounts<<<num_reads/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_reads, d_read_counts, num_reads);

    // sums all the values in read counts
    int read_counts_sum = thrust::reduce(thrust::device, dev_ptr, dev_ptr + std::pow(4,KMER),0);

    // calculate the prefix
    thrust::exclusive_scan(thrust::device, dev_ptr, dev_ptr + std::pow(4,KMER), d_read_prefix);

    // initialize an array that will contain the actual reads sorted by KMER
    int *d_read_chunk; hipMalloc(&d_read_chunk, read_counts_sum * sizeof(int));
    int *read_chunk = (int*) malloc(read_counts_sum * sizeof(int));

    // an array that will contain the IndexTable instances that for each KMER contains an index to the first read in read chunk and the amount of reads
    IndexTable *d_index_table; hipMalloc(&d_index_table, std::pow(4,KMER) * sizeof(IndexTable));hipMemset(d_index_table, 0, std::pow(4,KMER) * sizeof(IndexTable));
    IndexTable *index_table = (IndexTable*) malloc(std::pow(4,KMER) * sizeof(IndexTable));
    int *d_tmp_read_counts; hipMalloc(&d_tmp_read_counts, std::pow(4,KMER) * sizeof(int)); hipMemset(d_tmp_read_counts, 0, std::pow(4,KMER) * sizeof(int));

    // calculate index table
    computeIndexTable<<<num_reads/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_reads, d_read_counts, d_tmp_read_counts, d_read_chunk, d_read_prefix, d_index_table, num_reads);

    // delete no longer used arrays
    hipFree(d_read_counts);
    hipFree(d_tmp_read_counts);

    // for each read finds the closest read and its distance
    findClosest<<<num_reads/DIVIDE_DATA_BY,THREADS_PER_BLOCK>>>(d_reads,d_min_num, d_min_index, d_histograms,d_index_table,d_read_chunk, d_clock,d_counts);

    // clock that calculates average thread runtime, it was used for testing and debugging
    hipMemcpy(&clock, d_clock, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipMemcpy(counts, d_counts, num_reads * THREADS_PER_BLOCK * sizeof(long), hipMemcpyDeviceToHost);
    thrust::device_ptr<long> dev_counts_ptr(d_counts);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int clockRate = prop.clockRate * 1000;// its in KHz so * 1000 to Hz

    // copy results to CPU memory
    hipMemcpy(min_num, d_min_num, num_reads * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(min_index, d_min_index, num_reads * sizeof(int), hipMemcpyDeviceToHost);

    // print info
    std::cout<< "settings: " <<ETH << "," <<KMER << "," << DIVIDE_DATA_BY << "," << USED_READS_SIZE << "," <<static_cast<double> (clock) / (num_reads*THREADS_PER_BLOCK) / clockRate << "," <<static_cast<double> (thrust::reduce(thrust::device, dev_counts_ptr, dev_counts_ptr + num_reads * THREADS_PER_BLOCK,0l)) / (num_reads*THREADS_PER_BLOCK) << std::endl;

    // print results
    std::cout << "read index" << ","<< "closest read" << ","<< "edit distance" << std::endl;
    for(int i = 0; i < num_reads; i++){
        std::cout << i << ","<< min_index[i]<< ","<< min_num[i] << std::endl;
    }
    return 0;
}